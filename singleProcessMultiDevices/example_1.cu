//
// Example 1: Single Process, Single Thread, Multiple Devices
//

#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include <stdlib.h>
#include <vector>
#include <iostream>

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

__global__ void  init1(float *dptr,int i)
{
  int id = threadIdx.x;
  dptr[id] = id;
  printf("GPU: %d,dptr: %f\n",i,dptr[id]);
}

int main(int argc, char *argv[])
{
  std::cout << "================================================================"<<
               "\n    Executing " << argv[0] << " now!\n"<<
               "================================================================\n";
  ncclComm_t comms[2];

  // managing 2 devices
  int nDev = 2;
  const int size = 3;

  // std::vector<int> devs(nDev);
  // for (int i = 0; i < nDev; ++i)
  // {
  //   devs[i] = i;
  // }
  int devs[2] = {0, 1};

  // allocating and initializing device buffers
  float **sendbuff = (float **)malloc(nDev * sizeof(float *));
  float **recvbuff = (float **)malloc(nDev * sizeof(float *));
  float **hptr = (float **)malloc(nDev * sizeof(float *));
  // create nDev streams for ndev devices
  hipStream_t *s = (hipStream_t *)malloc(sizeof(hipStream_t) * nDev);

  for (int i = 0; i < nDev; ++i)
  {
    CUDACHECK(hipSetDevice(i));
    {
      // Device info
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, i);
      printf("\nDevice %d: \"%s\"%d,%d\n", i, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
    CUDACHECK(hipMalloc(sendbuff + i, size * sizeof(float)));
    CUDACHECK(hipMalloc(recvbuff + i, size * sizeof(float)));
    CUDACHECK(hipMemset(sendbuff[i], 1, size * sizeof(float)));
    CUDACHECK(hipMemset(recvbuff[i], 0, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(s + i));
    init1<<<1, size>>>(sendbuff[i], i);
  }

  NCCLCHECK(ncclCommInitAll(comms, nDev, devs));

  // calling NCCL communication API. Group API is required when
  // using multiple devices per thread
  NCCLCHECK(ncclGroupStart());
  for (int i = 0; i < nDev; ++i)
  {
    NCCLCHECK(ncclAllReduce((const void *)sendbuff[i],
                            (void *)recvbuff[i], size, ncclFloat, ncclSum,
                            comms[i], s[i]));
  }
  NCCLCHECK(ncclGroupEnd());

  // synchronizing on CUDA streams to wait for completion of NCCL operation
  for (int i = 0; i < nDev; ++i)
  {
    CUDACHECK(hipSetDevice(i));
    // it will stall host until all operations are done
    CUDACHECK(hipStreamSynchronize(s[i]));
  }

  // free device buffers
  for (int i = 0; i < nDev; ++i)
  {
    CUDACHECK(hipSetDevice(i));
    hptr[i] = (float *)malloc(size * sizeof(float));
    hipMemcpy(hptr[i], recvbuff[i], size * sizeof(float), hipMemcpyDeviceToHost);
    CUDACHECK(hipFree(sendbuff[i]));
    CUDACHECK(hipFree(recvbuff[i]));
  }

  // finalizing NCCL
  for (int i = 0; i < nDev; ++i)
  {
    ncclCommDestroy(comms[i]);
  }

  for (int i = 0; i < size; ++i)
  {
    for (int j = 0; j < nDev; ++j)
      std::cout<<"i= "<<i<<" "<<hptr[j][i]<<"\n";
    }
    free(hptr);
    printf("Success \n");
    return 0;
}