#include "hip/hip_runtime.h"
//
// Example 2: One Device Per Process Or Thread
//


#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include <iostream>
#include <printf.h>
#ifdef __linux
#include <unistd.h>
#endif

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);                      \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

static uint64_t getHostHash(const char *string) {
    // Based on DJB2, result = result * 33 + char
    uint64_t result = 5381;
    for (int c = 0; string[c] != '\0'; c++) {
        result = ((result << 5) + result) + string[c];
    }
    return result;
}

static void getHostName(char *hostname, int maxlen) {
    gethostname(hostname, maxlen);
    for (int i = 0; i < maxlen; i++) {
        if (hostname[i] == '.') {
            hostname[i] = '\0';
            return;
        }
    }
}

__global__ void  init(float *dptr,int myRank)
{
  int id = threadIdx.x;
  dptr[id] = id;
//   printf("kernel-myRank: %d id: %f\n",myRank,dptr[id]);
}

int main(int argc, char *argv[]) {
    {
        char host[256];
#ifdef __linux
        printf("PID %d on node %s is ready for attach\n",
               getpid(), host);
        fflush(stdout);
#endif
        if(argc!=2){
            std::cout<<"Please input a int,'0' means for debug,'1' means execute directly\n";
        }
        if (std::stoi(argv[1]) == 0)
        {
            {
                int i = 0;
                while (i == 0)
                {
                    i = 0;
                }
            }
        }
    }

    int size = 2;
    int myRank, nRanks, localRank = 0;
    // initializing MPI
    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));
    if (myRank == 0)
    {
        std::cout << "================================================================"
                  << "\n    Executing " << argv[0] << " now!\n"
                  << "================================================================\n";
    }
    std::cout<<"Rank: "<<myRank<<"\n";
    {
        int deviceCount = 0;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);
        for (int dev = 0; dev < deviceCount; ++dev)
        {
            hipSetDevice(dev);
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);
            printf("\nDevice %d: \"%s\"%d,%d\n", dev, deviceProp.name,deviceProp.major,deviceProp.minor);
        }
    }

    // calculating localRank based on hostname which is used in
    // selecting a GPU
    uint64_t hostHashs[nRanks];
    char hostname[1024];
    getHostName(hostname, 1024);
    hostHashs[myRank] = getHostHash(hostname);
    MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL,
                           hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
    for (int p = 0; p < nRanks; p++) {
        if (p == myRank) {
            break;
        }
        if (hostHashs[p] == hostHashs[myRank]) {
            localRank++;
        }
    }

    ncclUniqueId id;
    ncclComm_t comm;
    float *sendbuff;
    hipStream_t s;

    if (myRank == 0) {
        ncclGetUniqueId(&id);
    }
    MPICHECK(MPI_Bcast((void *) &id, sizeof(id), MPI_BYTE, 0,
                       MPI_COMM_WORLD));

    // picking a GPU based on localRank, allocate device buffers
    CUDACHECK(hipSetDevice(localRank));
    CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(&s));

    // call init kernel to init rank 0 sendbuff data
    if (myRank == 0)
        init<<<1, size>>>(sendbuff, myRank);

    // malloc host mem
    float *hptr = (float *)malloc(size * sizeof(float));
    hipMemcpy(hptr,sendbuff,size*sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "sendbuff-before-brdcast:\n";
    for (int i = 0; i < size; ++i)
    {
        std::cout << "myRank: " << myRank << " hptr["<<i<<"]: " << hptr[i] << "\n";
    }
    // initializing NCCL
    NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

    // communicating using NCCL
    NCCLCHECK(ncclBcast((void *)sendbuff,
                         size, ncclFloat, 0,
                         comm, s));
                         
    hipMemcpy(hptr,sendbuff,size*sizeof(float),hipMemcpyDeviceToHost);
    std::cout << "sendbuff-after-brdcast:\n";
    for (int i = 0; i < size; ++i)
    {
        std::cout << "myRank: " << myRank << " hptr["<<i<<"]: " << hptr[i] << "\n";
    }

    // completing NCCL operation by synchronizing on the CUDA stream
    CUDACHECK(hipStreamSynchronize(s));
    // free device buffers
    CUDACHECK(hipFree(sendbuff));

    // finalizing NCCL
    ncclCommDestroy(comm);

    // finalizing MPI
    MPICHECK(MPI_Finalize());
    free(hptr);
    printf("[MPI Rank %d] Success \n", myRank);
    // hipDeviceSynchronize();
    return 0;
}